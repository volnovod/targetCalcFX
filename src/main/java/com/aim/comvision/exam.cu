extern "C"

#include <hip/hip_runtime.h>
#include "stdio.h"

__global__ void example(int** A, int** B,int* C, int threads, int size){

    int i = threadIdx.x;
    C[i]=0;
    for(int k=0; k<threads; k++){
            C[i] += A[i][k] + B[i][k];
            printf(" A=%d C=%d  i=%d k=%d \n", A[k][i], C[k], i,k);
    }
    __syncthreads();
}