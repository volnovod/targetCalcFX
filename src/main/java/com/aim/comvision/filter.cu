#include "hip/hip_runtime.h"
extern "C"

#include "limits.h"
#include "stdlib.h"

void arrayToVector(int width, int height,int** array, int* vector){
    int counter = 0;
    for (int i = 0; i < height; ++i) {
        for (int j = 0; j < width; ++j) {
            vector[counter] = array[i][j];
            counter++;
        }
    }
}

int comparator (const void * a, const void * b)
{
    return ( *(int*)a - *(int*)b );
}

__global__ void filter(int width, int height, int** inputData){

    int i = threadIdx.x;
    int j = threadIdx.y;

    int** window;
    int vectorSize = 9;
    int* vectorFromArray;

    for (int i = 0; i < height-2; ++i) {
        for (int j = 0; j < width-2; ++j) {

            for (int k = 0; k < 3; ++k) {
                for (int l = 0; l < 3; ++l) {
                    window[k][l] =  inputData[i+k][j+l];
                }

            }

            arrayToVector(3,3, window, vectorFromArray);
            qsort(vectorFromArray, vectorSize, sizeof(int), comparator);
            inputData[i+1][j+1] = vectorFromArray[4];
        }
    }

}

