extern "C"

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#define MEDIAN_LENGTH 9

__global__ void filter(long *d_input_img, long*d_output_img, int d_iRows, int d_iCols){
    unsigned int row = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned int col = blockIdx.x*blockDim.x + threadIdx.x;
    long window[MEDIAN_LENGTH];

    if(col>=d_iCols || row>=d_iRows){
        return;
        }
    window[0]= (row==0||col==0) ? 0 :                 d_input_img[(row-1)*d_iCols+(col-1)];
    window[1]= (row==0) ? 0 :                         d_input_img[(row-1)*d_iCols+col];
    window[2]= (row==0||col==d_iCols-1) ? 0 :         d_input_img[(row-1)*d_iCols+(col+1)];
    window[3]= (col==0) ? 0 :                         d_input_img[row*d_iCols+(col-1)];
    window[4]=                                        d_input_img[row*d_iCols+col];
    window[5]= (col==d_iCols-1) ? 0 :                 d_input_img[row*d_iCols+(col+1)];
    window[6]= (row==d_iRows-1||col==0) ? 0 :         d_input_img[(row+1)*d_iCols+(col-1)];
    window[7]= (row==d_iRows-1) ? 0 :                 d_input_img[(row+1)*d_iCols+col];
    window[8]= (row==d_iRows-1||col==d_iCols-1) ? 0 : d_input_img[(row+1)*d_iCols+(col+1)];

    // Order elements
    for (unsigned int j=0; j<5; ++j)
    {
        // Find position of minimum element
        long temp = window[j];
        unsigned int  idx  = j;
        for (unsigned int l=j+1; l<9; ++l)
            if (window[l] < temp){ idx=l; temp = window[l];}
        // Put found minimum element in its place
        window[idx] = window[j];
        window[j] = temp;
    }

    d_output_img[row*d_iCols + col] = (window[4]);

}