
#include <hip/hip_runtime.h>
extern "C"

#define BLOCK_WIDTH 16
#define BLOCK_HEIGHT 16

__global__ void filter(long *Input_Image, long *Output_Image, int Image_Width, int Image_Height) {

    long surround[9];

    int iterator;

    const int x     = blockDim.x * blockIdx.x + threadIdx.x;
    const int y     = blockDim.y * blockIdx.y + threadIdx.y;

    if( (x >= (Image_Width - 1)) || (y >= Image_Height - 1) || (x == 0) || (y == 0)) return;

    // --- Fill array private to the threads
    iterator = 0;
    for (int r = x - 1; r <= x + 1; r++) {
        for (int c = y - 1; c <= y + 1; c++) {
            surround[iterator] = Input_Image[c*Image_Width+r];
            iterator++;
        }
    }

    // --- Sort private array to find the median using Bubble Short
    for (int i=0; i<5; ++i) {

        // --- Find the position of the minimum element
        int minval=i;
        for (int l=i+1; l<9; ++l) if (surround[l] < surround[minval]) minval=l;

        // --- Put found minimum element in its place
        long temp = surround[i];
        surround[i]=surround[minval];
        surround[minval]=temp;
    }

    // --- Pick the middle one
    Output_Image[(y*Image_Width)+x]=surround[4];

}