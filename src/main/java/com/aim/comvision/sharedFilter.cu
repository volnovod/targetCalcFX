
#include <hip/hip_runtime.h>
extern "C"

#define BLOCK_WIDTH 16
#define BLOCK_HEIGHT 16


__global__ void filter(int *Input_Image, int *Output_Image, int Image_Width, int Image_Height)
{
    const int tx_l = threadIdx.x;                           // --- Local thread x index
    const int ty_l = threadIdx.y;                           // --- Local thread y index

    const int tx_g = blockIdx.x * blockDim.x + tx_l;        // --- Global thread x index
    const int ty_g = blockIdx.y * blockDim.y + ty_l;        // --- Global thread y index

    __shared__ int smem[BLOCK_WIDTH+2][BLOCK_HEIGHT+2];

    // --- Fill the shared memory border with zeros
    if (tx_l == 0)                      smem[tx_l]  [ty_l+1]    = 0;    // --- left border
    else if (tx_l == BLOCK_WIDTH-1)     smem[tx_l+2][ty_l+1]    = 0;    // --- right border
    if (ty_l == 0) {                    smem[tx_l+1][ty_l]      = 0;    // --- upper border
        if (tx_l == 0)                  smem[tx_l]  [ty_l]      = 0;    // --- top-left corner
        else if (tx_l == BLOCK_WIDTH-1) smem[tx_l+2][ty_l]      = 0;    // --- top-right corner
        }   else if (ty_l == BLOCK_HEIGHT-1) {smem[tx_l+1][ty_l+2]  = 0;    // --- bottom border
        if (tx_l == 0)                  smem[tx_l]  [ty_l+2]    = 0;    // --- bottom-left corder
        else if (tx_l == BLOCK_WIDTH-1) smem[tx_l+2][ty_l+2]    = 0;    // --- bottom-right corner
    }

    // --- Fill shared memory
                                                                    smem[tx_l+1][ty_l+1] =                           Input_Image[ty_g*Image_Width + tx_g];      // --- center
    if ((tx_l == 0)&&((tx_g > 0)))                                      smem[tx_l]  [ty_l+1] = Input_Image[ty_g*Image_Width + tx_g-1];      // --- left border
    else if ((tx_l == BLOCK_WIDTH-1)&&(tx_g < Image_Width - 1))         smem[tx_l+2][ty_l+1] = Input_Image[ty_g*Image_Width + tx_g+1];      // --- right border
    if ((ty_l == 0)&&(ty_g > 0)) {                                      smem[tx_l+1][ty_l]   = Input_Image[(ty_g-1)*Image_Width + tx_g];    // --- upper border
            if ((tx_l == 0)&&((tx_g > 0)))                                  smem[tx_l]  [ty_l]   = Input_Image[(ty_g-1)*Image_Width + tx_g-1];  // --- top-left corner
            else if ((tx_l == BLOCK_WIDTH-1)&&(tx_g < Image_Width - 1))     smem[tx_l+2][ty_l]   = Input_Image[(ty_g-1)*Image_Width + tx_g+1];  // --- top-right corner
         } else if ((ty_l == BLOCK_HEIGHT-1)&&(ty_g < Image_Height - 1)) {  smem[tx_l+1][ty_l+2] = Input_Image[(ty_g+1)*Image_Width + tx_g];    // --- bottom border
         if ((tx_l == 0)&&((tx_g > 0)))                                 smem[tx_l]  [ty_l+2] = Input_Image[(ty_g-1)*Image_Width + tx_g-1];  // --- bottom-left corder
        else if ((tx_l == BLOCK_WIDTH-1)&&(tx_g < Image_Width - 1))     smem[tx_l+2][ty_l+2] = Input_Image[(ty_g+1)*Image_Width + tx_g+1];  // --- bottom-right corner
    }
    __syncthreads();

    // --- Pull the 3x3 window in a local array
    long v[9] = { smem[tx_l][ty_l],   smem[tx_l+1][ty_l],     smem[tx_l+2][ty_l],
                            smem[tx_l][ty_l+1], smem[tx_l+1][ty_l+1],   smem[tx_l+2][ty_l+1],
                            smem[tx_l][ty_l+2], smem[tx_l+1][ty_l+2],   smem[tx_l+2][ty_l+2] };

    // --- Bubble-sort
    for (int i = 0; i < 5; i++) {
        for (int j = i + 1; j < 9; j++) {
            if (v[i] > v[j]) { // swap?
                int tmp = v[i];
                v[i] = v[j];
                v[j] = tmp;
            }
         }
    }

    // --- Pick the middle one
    Output_Image[ty_g*Image_Width + tx_g] = v[4];
}