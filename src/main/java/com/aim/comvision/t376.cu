
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#define DCOLS 1024
#define DROWS 256

typedef struct {
  size_t step;
  size_t rows;
  size_t cols;
  unsigned char *data;
} mat;


// define the threads and grids for CUDA
#define BLOCK_ROWS 32
#define BLOCK_COLS 16

// define kernel dimensions
#define MEDIAN_LENGTH 9

// this is the error checking part for CUDA
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


__global__ void FilterKernel (unsigned char *d_input_img, unsigned char *d_output_img, int d_iRows, int d_iCols)

{

    unsigned int row = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned int col = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned char window[MEDIAN_LENGTH];

    if(col>=d_iCols || row>=d_iRows)
        return;

    window[0]= (row==0||col==0) ? 0 :                 d_input_img[(row-1)*d_iCols+(col-1)];
    window[1]= (row==0) ? 0 :                         d_input_img[(row-1)*d_iCols+col];
    window[2]= (row==0||col==d_iCols-1) ? 0 :         d_input_img[(row-1)*d_iCols+(col+1)];
    window[3]= (col==0) ? 0 :                         d_input_img[row*d_iCols+(col-1)];
    window[4]=                                        d_input_img[row*d_iCols+col];
    window[5]= (col==d_iCols-1) ? 0 :                 d_input_img[row*d_iCols+(col+1)];
    window[6]= (row==d_iRows-1||col==0) ? 0 :         d_input_img[(row+1)*d_iCols+(col-1)];
    window[7]= (row==d_iRows-1) ? 0 :                 d_input_img[(row+1)*d_iCols+col];
    window[8]= (row==d_iRows-1||col==d_iCols-1) ? 0 : d_input_img[(row+1)*d_iCols+(col+1)];

    // Order elements
    for (unsigned int j=0; j<5; ++j)
    {
        // Find position of minimum element
        unsigned char temp = window[j];
        unsigned int  idx  = j;
        for (unsigned int l=j+1; l<9; ++l)
            if (window[l] < temp){ idx=l; temp = window[l];}
        // Put found minimum element in its place
        window[idx] = window[j];
        window[j] = temp;
    }

    d_output_img[row*d_iCols + col] = (window[4]);

}

void take_input(const mat& input, const mat& output)
{

    unsigned char *device_input;
    unsigned char *device_output;

    size_t d_ipimgSize = input.step * input.rows;
    size_t d_opimgSize = output.step * output.rows;

    gpuErrchk( hipMalloc( (void**) &device_input, d_ipimgSize) );
    gpuErrchk( hipMalloc( (void**) &device_output, d_opimgSize) );

    gpuErrchk( hipMemcpy(device_input, input.data, d_ipimgSize, hipMemcpyHostToDevice) );

    dim3 Threads(BLOCK_COLS, BLOCK_ROWS);  // 512 threads per block
    dim3 Blocks((input.cols + Threads.x - 1)/Threads.x, (input.rows + Threads.y - 1)/Threads.y);

    //int check = (input.cols + Threads.x - 1)/Threads.x;
    //printf( "blockx %d", check);

    FilterKernel <<< Blocks, Threads >>> (device_input, device_output, input.rows, input.cols);
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipGetLastError());

    gpuErrchk( hipMemcpy(output.data, device_output, d_opimgSize, hipMemcpyDeviceToHost) );

    //printf( "num_rows_cuda %d", num_rows);
    //printf("\n");

    gpuErrchk(hipFree(device_input));
    gpuErrchk(hipFree(device_output));

}

int main(){
  mat input_im, output_im;
  input_im.rows  = DROWS;
  input_im.cols  = DCOLS;
  input_im.step  = input_im.cols;
  input_im.data  = (unsigned char *)malloc(input_im.step*input_im.rows);
  output_im.rows = DROWS;
  output_im.cols = DCOLS;
  output_im.step = input_im.cols;
  output_im.data = (unsigned char *)malloc(output_im.step*output_im.rows);

  for (int i = 0; i < DCOLS*DROWS; i++) {
    output_im.data[i] = 0;
    input_im.data[i] = 0;
    int temp = (i%DCOLS);
    if (temp == 5) input_im.data[i] = 20;
    if ((temp > 5) && (temp < 15)) input_im.data[i] = 40;
    if (temp == 15) input_im.data[i] = 20;
    }

  take_input(input_im, output_im);
  for (int i = 2*DCOLS; i < DCOLS*(DROWS-2); i++)
    if (input_im.data[i] != output_im.data[i]) {printf("mismatch at %d, input: %d, output: %d\n", i, (int)input_im.data[i], (int)output_im.data[i]); return 1;}
  printf("Success\n");
  return 0;
}